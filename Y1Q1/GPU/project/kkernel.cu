
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__device__ int findPosition(const int *a, int k, int b, int top){
	if(b < 0){
		return -1;
	}
	if(b > top){
		return -2;
	}
	for(int i = 0; i < k; i++){
		if(a[i] == b){
			return i;
		}	
	}
	return -3;
}

__global__ void parallelBFE(const int *x, const int *y, int *g, const int *a, const int *b, int n, int k, int M, int N, int E, int *N_DISKS){
	//int t = blockIdx.x * blockDim.x + threadIdx.x;
	int t = threadIdx.x;
	__shared__ int px[1000];
	__shared__ int py[1000];
	int j = 0;
	int h;

	// Center-Medium
	int cm = a[t];
	for(int i = b[t]; i < b[t + 1]; i++){
		px[j] = x[i];
		py[j] = y[i];
		j++;
	}
	h = j;

	// Left-Medium
	int lm;
	if(cm % M == 0){
		lm = -1;
	} else {
		lm = findPosition(a, k, cm - 1, M*N);
	}
	if(lm >= 0){
		for(int i = b[lm]; i < b[lm + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Right-Medium
	int rm;
	if(cm % M == M - 1){
		rm = -1;
	} else {
		rm = findPosition(a, k, cm + 1, M*N);
	}
	if(rm >= 0){
		for(int i = b[rm]; i < b[rm + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Center-Up
	int cu = cm - M;
	cu = findPosition(a, k, cu, M*N);
	if(cu >= 0){
		for(int i = b[cu]; i < b[cu + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Left-Up
	int lu;
	if(cm % M == 0){
		lu = -1;
	} else {
		lu = findPosition(a, k, cm - M - 1, M*N);
	}
	if(lu >= 0){
		for(int i = b[lu]; i < b[lu + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Right-Up
	int ru;
	if(cm % M == M - 1){
		ru = -1;
	} else {
		ru = findPosition(a, k, cm - M + 1, M*N);
	}
	if(ru >= 0){
		for(int i = b[ru]; i < b[ru + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Center-Down
	int cd = cm + M;
	cd = findPosition(a, k, cd, M*N);
	if(cd >= 0){
		for(int i = b[cd]; i < b[cd + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Left-Down
	int ld;
	if(cm % M == 0){
		ld = -1;
	} else {
		ld = findPosition(a, k, cm + M - 1, M*N);
	}
	if(ld >= 0){
		for(int i = b[ld]; i < b[ld + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	// Right-Down
	int rd;
	if(cm % M == M - 1){
		rd = -1;
	} else {
		rd = findPosition(a, k, cm + M + 1, M*N);
	}
	if(rd >= 0){
		for(int i = b[rd]; i < b[rd + 1]; i++){
			px[j] = x[i];
			py[j] = y[i];
			j++;
		}
	}
	//__syncthreads();
	N_DISKS[t] = j - h;
}


__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float *C) {
    // Declaring the variables in shared memory...
	__shared__ float A_s[TILE_SIZE][TILE_SIZE];
	__shared__ float B_s[TILE_SIZE][TILE_SIZE];

	// Finding the coordinates for the current thread...
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = blockIdx.x * blockDim.x + tx;
	int row = blockIdx.y * blockDim.y + ty;

	float sum = 0.0f;

	for(int i = 0; i < ((k - 1) / TILE_SIZE) + 1; ++i){
		// Validation in the case the thread tries to write in share 
		// memory a value outside the dimensions of matrix A...
		if(row < m && (i * TILE_SIZE + tx) < k){
			A_s[ty][tx] = A[(row * k) + (i * TILE_SIZE + tx)];
		} else {
			// In that case, just write a 0 which will no affect 
			// the computation...
			A_s[ty][tx] = 0.0f;
		}
		// Similar validation for B...
		if((i * TILE_SIZE + ty) < k && col < n){
			B_s[ty][tx] = B[((i * TILE_SIZE + ty) * n) + col];
		} else {
			B_s[ty][tx] = 0.0f;
		}
		// Wait for all the threads to write in share memory
		__syncthreads();

		// Compute the multiplication on the tile...
		for(int j = 0; j < TILE_SIZE; ++j){
			sum += A_s[ty][j] * B_s[j][tx];
		}
		// Wait to finish before to go ahead with the next phase...
		__syncthreads();
	}
	// Write the final result in C just if it is inside of the valid 
	// dimensions... 
	if(row < m && col < n){
		C[row * n + col] = sum;
	}
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    const unsigned int BLOCK_SIZE = TILE_SIZE;

    // Initialize thread block and kernel grid dimensions
    const dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
    const dim3 dim_grid(((n - 1) / BLOCK_SIZE) + 1, ((m - 1) / BLOCK_SIZE) + 1, 1);

    // Calling the kernel with the above-mentioned setting... 
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);
}